#include <cstdio>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

using namespace std;

int main(){
	/* Dimensão da matriz Hermitiana/Laplaciana
	 */
	int n = 5000;

	/* Tempo
	 */
	float t = 1;

	/* Matriz Hermitiana/Laplaciana com
	 * dimensões NxN e seus valores zerados.
	 */
	vector<hipFloatComplex> L(n * n, make_hipFloatComplex(.0f, .0f));

	/* Define os valores da matriz Hermitiana/Laplaciana
	 * de forma simétrica correspondente as adjacências
	 * de um grafo.
	 */
	for(int i = 0; i < n; i++){
		for(int j = i + 1; j < n; j++){
			int value = rand() % 2 - 1;
			L[i * n + j] = make_hipFloatComplex(value, .0f);
			L[j * n + i] = make_hipFloatComplex(value, .0f);
		}
	}

	/* Define os valores da diagonal principal
	 * da matriz Hermitiana/Laplaciana correspondente
	 * ao grau do vértice.
	 * 
	 * Da para otimizar essa definição sem ter que percorrer
	 * todos os valores, apenas os elementos acima ou abaixo
	 * da diagonal
	 */
	for(int i = 0; i < n; i++){
		float degree = .0f;

		for(int j = 0; j < n; j++){
			degree += hipCabsf(L[i * n + j]);
		}
		
		L[i * n + i] = make_hipFloatComplex(degree, .0f);
	}

	/* Aloca espaço na memória da placa de video para
	 * armazenar os autovalores e autovetores calculados.
	 */
	hipComplex *dEigenvectors = nullptr;
	hipComplex *dWork         = nullptr; 
	float *dEigenvalues      = nullptr;
	int   *dInfo             = nullptr;

	hipMalloc(&dEigenvectors, sizeof(hipComplex) * n * n);
  	hipMalloc(&dEigenvalues, sizeof(float) * n);
  	hipMalloc(&dInfo, sizeof(int));
  	hipMemcpy(dEigenvectors, L.data(), sizeof(hipComplex) * n * n, hipMemcpyHostToDevice);

	hipsolverHandle_t handle_cusolver = nullptr;
	hipsolverDnCreate(&handle_cusolver);

	/* Calcula espaço necessário na memória da placa 
	 * de video para a realização dos calculos.
	 */
	int lwork = 0;
	hipsolverDnCheevd_bufferSize(
		handle_cusolver,
		HIPSOLVER_EIG_MODE_VECTOR,
		HIPBLAS_FILL_MODE_LOWER,
		n,
		dEigenvectors,
		n,
		dEigenvalues,
		&lwork
	);

	/* Aloca espaço na memória da placa de vídeo para
	 * a realização dos calculos.
	 */
	hipMalloc(&dWork, sizeof(hipComplex)*lwork);
	
	/* Realiza o calculo dos autovalores e autovetores 
	 * da matriz Hermitiana/Laplaciana.
	 */
	hipsolverDnCheevd(
		handle_cusolver,
		HIPSOLVER_EIG_MODE_VECTOR,
		HIPBLAS_FILL_MODE_LOWER,
		n,
		dEigenvectors,
		n,
		dEigenvalues,
		dWork,
		lwork,
		dInfo
	);

	int info = 0;
  	hipMemcpy(&info, dInfo, sizeof(int), hipMemcpyDeviceToHost);

	if (info != 0) {
    	fprintf(stderr, "Cheevd falhou: info=%d\n", info);
    	return 1;
  	}

	/* Copia os autovalores e autovetores calculados da memória
	 * da placa de vídeo para a memória RAM. 
	 */
	vector<float> hEigenvalues(n);
	vector<hipComplex> hEigenvectors(n * n);
        hipMemcpy(hEigenvalues.data(), dEigenvalues, sizeof(float) * n, hipMemcpyDeviceToHost);
	hipMemcpy(hEigenvectors.data(), dEigenvectors, sizeof(hipComplex) * n * n, hipMemcpyDeviceToHost);

	/* Aloca algumas variavéis para calculo da evolução temporal
	 */
	hipblasHandle_t handle_cublas = nullptr;
	hipblasCreate(&handle_cublas);

	vector<hipComplex> psi_t(n, make_hipFloatComplex(.0f, .0f));
	vector<hipComplex> psi_0(n, make_hipFloatComplex(.0f, .0f));
	psi_0[0] = make_hipFloatComplex(1.f, .0f);

	hipComplex *dpsi_t = nullptr;
	hipComplex *dpsi_0 = nullptr;
	
	hipMalloc(&dpsi_t, sizeof(hipComplex) * n);
  	hipMalloc(&dpsi_0, sizeof(hipComplex) * n);

	hipMemcpy(dpsi_t, psi_t.data(), sizeof(hipComplex) * n, hipMemcpyHostToDevice);
	hipMemcpy(dpsi_0, psi_0.data(), sizeof(hipComplex) * n, hipMemcpyHostToDevice);


	for(int k = 0; k < n; k++){
		hipComplex* eigenvector_k = dEigenvectors + k * n;
		hipComplex dot;
		
		/* Calcula e^{-i λ_k t}.
		 */
		float theta = -hEigenvalues[k] * t;
		hipFloatComplex phase = make_hipFloatComplex(cosf(theta), sinf(theta));

		/* Calcula <Ø_k|ψ_0>.
		 */
		hipblasCdotc(handle_cublas, n, eigenvector_k, 1, dpsi_0, 1, &dot);

		/* Calcula e^{-i λ_k t} <Ø_k|ψ_0>.
		 */
		hipComplex coeff = hipCmulf(dot, phase);

		/* Calcula e^{-i λ_k t} <Ø_k|ψ_0> |Ø_k> e soma o resultado
		 * ao ψ(t).
		 */
		hipblasCaxpy(handle_cublas, n, &coeff, eigenvector_k, 1, dpsi_t, 1);
	}

    hipMemcpy(psi_t.data(), dpsi_t, sizeof(hipComplex)*n, hipMemcpyDeviceToHost);

    printf("ψ(%.3f):\n", t);

    for(int i = 0; i < n; i++){
        float re   = hipCrealf(psi_t[i]);
        float im   = hipCimagf(psi_t[i]);
        float prob = re * re + im * im;

        printf("  node %d: (%.4f, %.4f), |ψ|² = %.4f\n", i, re, im, prob);
    }

	/* Libera espaço de memória alocado durante
	 * a execução do programa.
	 */
	hipsolverDnDestroy(handle_cusolver);
	hipblasDestroy(handle_cublas);
	hipFree(dpsi_t); 
	hipFree(dpsi_0); 
	hipFree(dWork); 
	hipFree(dInfo); 
	hipFree(dEigenvectors); 
	hipFree(dEigenvalues);
	
	return 0;
}
